#include "hip/hip_runtime.h"
__global__ void
invert(const int length, const double *a, hipDoubleComplex *b) {
    if (threadIdx.x < length && fabs(a[threadIdx.x]) > 1.0e-10)
        b[threadIdx.x + threadIdx.x * length] = make_hipDoubleComplex(1 / a[threadIdx.x], 0);
}

__global__ void
get_power(const hipDoubleComplex *response, double *power_output, int &time_index) {
    power_output[time_index] = hipCreal(hipCmul(*response, hipConj(*response)));
    time_index++;
}

void cudaErrorChk(char* process) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Error during " << process << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}