#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addGPU(unsigned int n, const float *x, const float *y, float *z) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = index; i < n; i += stride)
        z[i] = x[i] + y[i];
}

int main() {
    //error checking
    hipError_t err = hipSuccess;

    // init array size
    unsigned int *n;
    err = hipMallocManaged(&n, sizeof(unsigned int));
    if (err != hipSuccess) {
        std::cout << "Failed to allocate n on device: " << err << std::endl;
    }
    n[0] = 1 << 20;

    //init arrays
    float *x, *y, *z;
    hipMallocManaged(&x, n[0] * sizeof(float));
    hipMallocManaged(&y, n[0] * sizeof(float));
    hipMallocManaged(&z, n[0] * sizeof(float));
    for (int i = 0; i < n[0]; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    unsigned int blockSize = 256;
    unsigned int NumBlocks = (n[0] + blockSize - 1) / blockSize;
    long start = clock();
    addGPU<<<NumBlocks, blockSize>>>(n[0], x, y, z);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Failed to launch kernel: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cout << "Failed to sync threads: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    long duration = clock() - start;


    double maxError = 0.0f;
    for (int i = 0; i < n[0]; i++) {
        maxError = fmax(maxError, fabs(z[i] - 3.0f));
    }

    std::cout << "Maximum error is: " << maxError << ", completed in " << duration << " µs" << std::endl;

    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipFree(n);
    return 0;
}
